///ETETechnology cuda assignment
#include <hip/hip_runtime.h>

#include <iostream>
#include <vector>
#include <algorithm>
#include <ctime>
#include <thread>
#include <chrono>

using namespace std;

// Calculate the positions of the runners in parallel
__global__ void race(float *positions, float *speeds, int n) {
    int i = blockDim.x * blockIdx.x + threadIdx.x;
    if (i < n) {
        positions[i] += speeds[i];
    }
}

struct Runner {
    int id;
    float position;

    bool operator<(const Runner& other) const {
        return position > other.position;
    }
};

int main() {
    srand(time(0));
    const int n = 100;
    vector<Runner> runners(n);
    float speeds[n];
    float *d_positions, *d_speeds;

    hipMalloc((void **)&d_positions, n * sizeof(float));
    hipMalloc((void **)&d_speeds, n * sizeof(float));

    // Initialize runners and their speeds
    for (int i = 0; i < n; ++i) {
        runners[i].id = i + 1;  
        runners[i].position = 0.0f;
        speeds[i] = 1.0f + rand() % 5;
    }

    hipMemcpy(d_positions, &runners[0].position, n * sizeof(float), hipMemcpyHostToDevice);
    hipMemcpy(d_speeds, speeds, n * sizeof(float), hipMemcpyHostToDevice);

    int blockSize = 256;
    int gridSize = (n + blockSize - 1) / blockSize;
    bool finished = false;

    // Run the race
    while (!finished) {
        race<<<gridSize, blockSize>>>(d_positions, d_speeds, n);
        hipMemcpy(&runners[0].position, d_positions, n * sizeof(float), hipMemcpyDeviceToHost);

        for (const auto &runner : runners) {
            if (runner.position >= 100.0f) {    
                finished = true;
                break;
            }
        }

        std::this_thread::sleep_for(std::chrono::seconds(1));
    }

    // Sort runners by their positions
    sort(runners.begin(), runners.end());

    // Display the results
    for (int i = 0; i < n; ++i) {
        cout << "Runner " << runners[i].id << " finished in position " << i + 1 << " with a distance of " << runners[i].position << " meters." << endl;
    }

    hipFree(d_positions);
    hipFree(d_speeds);

    return 0;
}
